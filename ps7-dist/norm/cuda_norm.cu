#include "hip/hip_runtime.h"
#include "Timer.hpp"
#include <iostream>
#include <math.h>
#include <vector>


__global__
void sq(int n, float *x, float *y){
  /* Write me -- kernel */
  extern __shared__ float sdata[256];

  size_t tid = threadIdx.x;
  size_t i = blockIdx.x*(blockDim.x*2) + threadIdx.x;
  sdata[tid] = x[i] * x[i];
  __syncthreads();

  for (size_t s=1; s < blockDim.x; s *= 2) {
    if (tid % (2*s) == 0) {
    sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  if (tid == 0)
    y[blockIdx.x] = sdata[0];
}


int main(int argc, char* argv[]) {

  size_t exponent           = 27;
  size_t num_trips          = 1;

  if (argc >= 2) exponent   = std::stol(argv[1]);
  if (argc >= 3) num_trips  = std::stol(argv[2]);

  size_t N = 1 << exponent;
  int block_size = 256;
  int num_blocks = (N + block_size - 1) / block_size;

  float *x = nullptr, *y = nullptr;

  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, num_blocks*sizeof(float));

  for (int i = 0; i < N; i++) {
    x[i] = i;
  }

  DEF_TIMER(cuda_norm);
  START_TIMER(cuda_norm);
  hipDeviceSynchronize();

  for (size_t i = 0; i < num_trips; ++i) {
    /* write me -- launch sq kernel */
    sq<<<num_blocks, block_size>>>(N, x, y);
    hipDeviceSynchronize();
  }
  /* write me: final step, copy out values from y and add on cpu */  
  float norm = 0.0;
  for (size_t i = 0; i < num_blocks; ++i) {
    norm += y[i];
  }
  norm = sqrt(norm);

  double cuda_time = STOP_TIMER_QUIETLY(cuda_norm);
  std::cout << exponent << "\t" << num_trips << "\t" << cuda_time << std::endl;

  hipFree(x); hipFree(y);
  
  return 0;
}